#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

__global__ void kernel_load_mode_vec(
            int n_particles,
            float *d_modes, 
            int *d_type,
            hipComplex *d_mode_vec)
    {
    int n = blockIdx.x*blockDim.x + threadIdx.x;

    if (n >= n_particles)
        return;

    int type = d_type[n];
    d_mode_vec[n] = make_hipComplex(d_modes[type],0.0f);
    }

__global__ void kernel_load_matrix(
            int n_wave,
            hipComplex *exp_matrix,
            int pitch,
            float3 *pos,
            float3 *wave_vectors)
    {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n_wave)
        return;

    int j = blockIdx.y;

    float3 q = wave_vectors[i];
    float3 p = pos[j];
    float dotproduct = q.x * p.x + q.y * p.y + q.z * p.z;
    // store in column-major format
    exp_matrix[j * pitch + i] = make_hipComplex(cosf(dotproduct),
                                                   sinf(dotproduct));
    }

__global__ void kernel_calculate_norms(hipComplex* fourier_mode_vec,
                                       float *sq_vec,
                                       int n_wave,
                                       float V)
    {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_wave)
        return;
    hipComplex fourier_mode = fourier_mode_vec[i];
    float normsq = fourier_mode.x * fourier_mode.x + fourier_mode.y * fourier_mode.y;
    sq_vec[i] = normsq/V;
    }

int gpu_sample_structure_factor(int n_wave,
                                 float3 *h_wave_vectors,
                                 unsigned int n_particles,
                                 float3 *h_pos,
                                 int *h_types,
                                 int n_type,   
                                 int n_mode,
                                 float *h_modes,
                                 float *h_sq,
                                 float V
                                 ) 
    {
    float3* d_wave_vectors;
    float3* d_pos;
    int *d_type;
    float *d_modes;
    hipComplex *d_mode_vec;
    hipComplex *d_fourier_mode_vec;
    float *d_sq_vec;

    hipError_t cudaStatus;

    hipMalloc(&d_wave_vectors, sizeof(float3)*n_wave);
    hipMemcpy(d_wave_vectors, h_wave_vectors, sizeof(float3)*n_wave, hipMemcpyHostToDevice);

    hipMalloc(&d_pos, sizeof(float3)*n_particles);
    hipMemcpy(d_pos, h_pos, sizeof(float3)*n_particles, hipMemcpyHostToDevice);

    hipMalloc(&d_type, sizeof(int)*n_particles);
    hipMemcpy(d_type, h_types, sizeof(int)*n_particles, hipMemcpyHostToDevice);

    hipMalloc(&d_modes, sizeof(float)*n_type*n_mode);
    hipMemcpy(d_modes, h_modes,sizeof(float)*n_type*n_mode,hipMemcpyHostToDevice);

    hipMalloc(&d_mode_vec, sizeof(hipComplex)*n_particles);
    hipComplex *d_exp_matrix;
    size_t pitch;

    hipMallocPitch((void **)&d_exp_matrix, &pitch, (size_t) (sizeof(hipComplex)*n_wave), (size_t)n_particles);
    pitch/=sizeof(hipComplex);
    hipMalloc(&d_fourier_mode_vec, sizeof(hipComplex)*n_wave);
    hipMalloc(&d_sq_vec, sizeof(float)*n_wave*n_mode);


    // initialize cuBLAS
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if ( stat != HIPBLAS_STATUS_SUCCESS )
        {
        printf("CUBLAS Error %d\n", stat);
        return 1;
        }

    for (int i = 0; i < n_mode; i++)
        {
        // load mode vector
        int block_size = 512;

        kernel_load_mode_vec<<<n_particles/block_size + 1, block_size>>>(
            n_particles,
            d_modes + i*n_type, 
            d_type,
            d_mode_vec);

        if (cudaStatus = hipGetLastError())
            {
            printf("CUDA ERROR: %s\n", hipGetErrorString(cudaStatus));
            return 1;
            }

        // load exponential factor matrix
        dim3 dimGrid(n_wave/block_size + 1,n_particles);
        dim3 dimBlock(block_size,1);
        kernel_load_matrix<<<dimGrid, dimBlock>>>(
            n_wave,
            d_exp_matrix,
            pitch,
            d_pos,
            d_wave_vectors);

        if (cudaStatus = hipGetLastError())
            {
            printf("CUDA ERROR: %s\n", hipGetErrorString(cudaStatus));
            return 1;
            }

        // matrix multiplication of exp_matrix with mode_vec
        hipComplex alpha = make_hipComplex(1.0f,0.0f);

        hipComplex beta = make_hipComplex(0.0f,0.0f);

        stat = hipblasCgemv(handle,
                    HIPBLAS_OP_N,
                    n_wave, 
                    n_particles,
                    &alpha,
                    d_exp_matrix,
                    pitch,
                    d_mode_vec, 1,
                    &beta,
                    d_fourier_mode_vec, 1);

        if ( stat != HIPBLAS_STATUS_SUCCESS )
            {
            printf("CUBLAS Error %d\n", stat);
            return 1;
            }

        // calculate norms of the entries of sq_vec
        
        kernel_calculate_norms<<<n_wave/block_size + 1, block_size>>>(d_fourier_mode_vec,
                                                                      d_sq_vec + i*n_wave,
                                                                      n_wave,
                                                                      V);

        if (cudaStatus = hipGetLastError())
            {
            printf("CUDA ERROR: %s\n", hipGetErrorString(cudaStatus));
            return 1;
            }


        } // end loop over modes

    stat = hipblasDestroy ( handle ) ;
    if ( stat != HIPBLAS_STATUS_SUCCESS )
        {
        printf("CUBLAS Error %d\n", stat);
        return 1;
        }
    // copy back structure factors
    hipMemcpy(h_sq, d_sq_vec, n_wave*n_mode*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_wave_vectors);
    hipFree(d_pos);
    hipFree(d_type);
    hipFree(d_modes);
    hipFree(d_mode_vec);
    hipFree(d_exp_matrix);
    hipFree(d_fourier_mode_vec);
    hipFree(d_sq_vec);

    return 0;
    }
